#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/fft_op.h"

#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"


namespace caffe2 {
namespace {
__global__ void GetComplexKernel(const int N, const hipfftReal* Real_, 
        const hipfftReal* Imag_, hipfftComplex* Complex_) {
    CUDA_1D_KERNEL_LOOP(i, N) {
        Complex_[i].x = Real_[i]; 
        Complex_[i].y = Imag_[i]; 
    }
}

__global__ void GetRealAndImageKernel(const int N, const hipfftComplex* Complex_, 
        hipfftReal* Real_, hipfftReal* Imag_) {
    CUDA_1D_KERNEL_LOOP(i, N) {
        Real_[i] = Complex_[i].x;
        Imag_[i] = Complex_[i].y;
    }
}
}  // namespace

template <>
bool FFTOp<float, HIPContext>::RunOnDevice() {
  /// schema checking
  auto& XReal = Input(0);
  auto& XImag = Input(1);
  auto* YReal = Output(0);
  auto* YImag = Output(1);
  CAFFE_ENFORCE_EQ(XReal.ndim(), 2);
  CAFFE_ENFORCE_EQ(XReal.ndim(), XImag.ndim());

  const auto canonical_axis_input = XReal.canonical_axis_index(1);
  int N = XReal.size_to_dim(canonical_axis_input);
  int D = XReal.size_from_dim(canonical_axis_input);
  YReal->ResizeLike(XReal);
  YImag->ResizeLike(XImag);

  /// computing
  // memory allocation
  hipfftComplex *XComplex; 
  CUDA_ENFORCE(hipMalloc(&XComplex, N * D *  sizeof(hipfftComplex)));
  hipfftComplex *YComplex; 
  CUDA_ENFORCE(hipMalloc(&YComplex, N * D *  sizeof(hipfftComplex)));

  // merge input real and imag part for hipfftExecC2C transformation
  GetComplexKernel<<<
      CAFFE_GET_BLOCKS(XReal.size()),
      CAFFE_CUDA_NUM_THREADS, 
      0, context_.hip_stream()>>>( 
          XReal.size(),
          XReal.data<float>(),
          XImag.data<float>(),
          XComplex);

  // batched 1D FFT forward
  hipfftHandle handle;
  int rank = 1;                       // 1D FFTs
  int n[] = {D};                      // Size of the Fourier transform
  int istride = 1, ostride = 1;       // Distance between two successive input/output elements
  int idist = D, odist = D;           // Distance between batches
  int inembed[] = { 0 };              // Input size with pitch (ignored for 1D transforms)
  int onembed[] = { 0 };              // Output size with pitch (ignored for 1D transforms)
  int batch = N;                      // Number of batched executions
  hipfftPlanMany( 
      &handle, rank, n, 
      inembed, istride, idist, 
      onembed, ostride, odist, 
      HIPFFT_C2C, 
      batch);
  hipfftExecC2C(handle, XComplex, YComplex, HIPFFT_FORWARD);
  hipfftDestroy(handle);

  // split real and image part of FFT output
  GetRealAndImageKernel<<<
      CAFFE_GET_BLOCKS(YReal->size()), 
      CAFFE_CUDA_NUM_THREADS, 
      0, context_.hip_stream()>>>( 
              YReal->size(),
              YComplex,
              YReal->mutable_data<float>(), 
              YImag->mutable_data<float>());

  // free tmp GPU memory
  CUDA_ENFORCE(hipFree(XComplex));
  CUDA_ENFORCE(hipFree(YComplex));

  return true;
}

template <>
bool FFTGradientOp<float, HIPContext>::RunOnDevice() {
  /// schema checking
  auto& dYReal = Input(0);
  auto& dYImag = Input(1);
  auto* dXReal = Output(0);
  auto* dXImag = Output(1);
  CAFFE_ENFORCE_EQ(dYReal.ndim(), 2);
  CAFFE_ENFORCE_EQ(dYReal.ndim(), dYImag.ndim());

  const auto canonical_axis_input = dYReal.canonical_axis_index(1);
  int N = dYReal.size_to_dim(canonical_axis_input);
  int D = dYReal.size_from_dim(canonical_axis_input);
  dXReal->ResizeLike(dYReal);
  dXImag->ResizeLike(dYImag);

  /// computing
  // memory allocation
  hipfftComplex *dYComplex; 
  CUDA_ENFORCE(hipMalloc(&dYComplex, N * D *  sizeof(hipfftComplex)));
  hipfftComplex *dXComplex; 
  CUDA_ENFORCE(hipMalloc(&dXComplex, N * D *  sizeof(hipfftComplex)));

  // merge input real and imag part for hipfftExecC2C transformation
  GetComplexKernel<<<
      CAFFE_GET_BLOCKS(dYReal.size()),
      CAFFE_CUDA_NUM_THREADS, 
      0, context_.hip_stream()>>>( 
          dYReal.size(),
          dYReal.data<float>(),
          dYImag.data<float>(),
          dYComplex);

  // batched 1D FFT forward
  hipfftHandle handle;
  int rank = 1;                       // 1D FFTs
  int n[] = {D};                      // Size of the Fourier transform
  int istride = 1, ostride = 1;       // Distance between two successive input/output elements
  int idist = D, odist = D;           // Distance between batches
  int inembed[] = { 0 };              // Input size with pitch (ignored for 1D transforms)
  int onembed[] = { 0 };              // Output size with pitch (ignored for 1D transforms)
  int batch = N;                      // Number of batched executions
  hipfftPlanMany( 
      &handle, rank, n, 
      inembed, istride, idist, 
      onembed, ostride, odist, 
      HIPFFT_C2C, 
      batch);
  hipfftExecC2C(handle, dYComplex, dXComplex, HIPFFT_FORWARD);
  hipfftDestroy(handle);

  // split real and image part of FFT output
  GetRealAndImageKernel<<<
      CAFFE_GET_BLOCKS(dXReal->size()), 
      CAFFE_CUDA_NUM_THREADS, 
      0, context_.hip_stream()>>>( 
              dXReal->size(),
              dXComplex,
              dXReal->mutable_data<float>(), 
              dXImag->mutable_data<float>());

  // free tmp GPU memory
  CUDA_ENFORCE(hipFree(dYComplex));
  CUDA_ENFORCE(hipFree(dXComplex));

  return true;
}

REGISTER_HIP_OPERATOR(FFT, FFTOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(FFTGradient, FFTGradientOp<float, HIPContext>);
}  // namespace caffe2
